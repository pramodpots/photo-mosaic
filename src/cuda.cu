#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "cuda.cuh"

#include <cstring>

#include "helper.h"
#include "hip/hip_runtime.h"
#include ""

///
/// Algorithm storage
///
// Host copy of input image
Image cuda_input_image;
// Host copy of image tiles in each dimension
unsigned int cuda_TILES_X, cuda_TILES_Y, cuda_input_image_width, cuda_input_image_height, cuda_input_image_channels;

// Pointer to device buffer for calculating the sum of each tile mosaic, this must be passed to a kernel to be used on device
unsigned long long* d_mosaic_sum;
// Pointer to device buffer for storing the output pixels of each tile, this must be passed to a kernel to be used on device
unsigned char* d_mosaic_value;
// Pointer to device image data buffer, for storing the input image, this must be passed to a kernel to be used on device
unsigned char* d_input_image_data;
// Pointer to device image data buffer, for storing the output image data, this must be passed to a kernel to be used on device
unsigned char* d_output_image_data;
// Pointer to device buffer for the global pixel average sum, this must be passed to a kernel to be used on device
unsigned long long* d_global_pixel_sum;

void cuda_begin(const Image *input_image) {
    // These are suggested CUDA memory allocations that match the CPU implementation
    // If you would prefer, you can rewrite this function (and cuda_end()) to suit your preference

    cuda_TILES_X = input_image->width / TILE_SIZE;
    cuda_TILES_Y = input_image->height / TILE_SIZE;

    cuda_input_image_width = input_image->width;
    cuda_input_image_height = input_image->height;
    cuda_input_image_channels = input_image->channels;

    // Allocate buffer for calculating the sum of each tile mosaic
    CUDA_CALL(hipMalloc(&d_mosaic_sum, cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned long long)));

    // Allocate buffer for storing the output pixel value of each tile
    CUDA_CALL(hipMalloc(&d_mosaic_value, cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned char)));

    const size_t image_data_size = input_image->width * input_image->height * input_image->channels * sizeof(unsigned char);
    // Allocate copy of input image
    cuda_input_image = *input_image;
    cuda_input_image.data = (unsigned char*)malloc(image_data_size);
    memcpy(cuda_input_image.data, input_image->data, image_data_size);

    // Allocate and fill device buffer for storing input image data
    CUDA_CALL(hipMalloc(&d_input_image_data, image_data_size));
    CUDA_CALL(hipMemcpy(d_input_image_data, input_image->data, image_data_size, hipMemcpyHostToDevice));

    // Allocate device buffer for storing output image data
    CUDA_CALL(hipMalloc(&d_output_image_data, image_data_size));

    // Allocate and zero buffer for calculation global pixel average
    CUDA_CALL(hipMalloc(&d_global_pixel_sum, input_image->channels * sizeof(unsigned long long)));
}

__global__ void tile_sum_CUDA(unsigned char* d_input_image_data, unsigned long long* d_mosaic_sum, unsigned int cuda_TILES_X, unsigned int cuda_TILES_Y, unsigned int cuda_input_image_width, unsigned int cuda_input_image_height, unsigned int cuda_input_image_channels) {
    // Block index
    int t_x = threadIdx.x + blockIdx.x * blockDim.x;
    int t_y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_x + t_y * blockDim.x * gridDim.x;

    const unsigned int tile_index = (t_y * cuda_TILES_X + t_x) * cuda_input_image_channels;
    const unsigned int tile_offset = (t_y * cuda_TILES_Y * TILE_SIZE * TILE_SIZE + t_x * TILE_SIZE) * cuda_input_image_channels;
    //const unsigned char pixel_2 = d_input_image_data[offset];
    atomicAdd(&d_mosaic_sum[tile_index+0], d_input_image_data[offset+0]);
    atomicAdd(&d_mosaic_sum[tile_index + 1], d_input_image_data[offset + 1]);
    atomicAdd(&d_mosaic_sum[tile_index + 2], d_input_image_data[offset + 2]);
    // For each pixel within the tile
    //for (int p_x = 0; p_x < TILE_SIZE; ++p_x) {
    //    for (int p_y = 0; p_y < TILE_SIZE; ++p_y) {
    //        // For each colour channel
    //        const unsigned int pixel_offset = (p_y * cuda_input_image_width + p_x) * cuda_input_image_channels;
    //        for (int ch = 0; ch < cuda_input_image_channels; ++ch) {
    //            // Load pixel
    //            const unsigned char pixel = d_input_image_data[tile_offset + pixel_offset + ch];
    //            const unsigned char pixel_2 = d_input_image_data[offset];
    //            d_mosaic_sum[tile_index + ch] += pixel;
    //            /*d_mosaic_sum[tile_index + ch] += pixel; 
    //            atomicAdd(&d_mosaic_sum[tile_index + ch], pixel);*/
    //        }
    //    }
    //}


    //for (unsigned int t_x = 0; t_x < cuda_TILES_X; ++t_x) {
    //    for (unsigned int t_y = 0; t_y < cuda_TILES_Y; ++t_y) {
    //        const unsigned int tile_index = (t_y * cuda_TILES_X + t_x) * cuda_input_image_channels;
    //        const unsigned int tile_offset = (t_y * cuda_TILES_X * TILE_SIZE * TILE_SIZE + t_x * TILE_SIZE) * cuda_input_image_channels;
    //        // For each pixel within the tile
    //        for (int p_x = 0; p_x < TILE_SIZE; ++p_x) {
    //            for (int p_y = 0; p_y < TILE_SIZE; ++p_y) {
    //                // For each colour channel
    //                const unsigned int pixel_offset = (p_y * cuda_input_image_width + p_x) * cuda_input_image_channels;
    //                for (int ch = 0; ch < cuda_input_image_channels; ++ch) {
    //                    // Load pixel
    //                    const unsigned char pixel = d_input_image_data[tile_offset + pixel_offset + ch];
    //                    d_mosaic_sum[tile_index + ch] += pixel;
    //                }
    //            }
    //        }
    //    }
    //}
}

void print_device_arch() {
    int major = 0;
    int minor = 0;

    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, 0);
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, 0);
    //Compute the arch integer value.
    int arch = (10 * major) + minor;
    printf("Device arch: %d\n", arch);
}
void cuda_stage1() {
    dim3 blocksPerGrid(cuda_TILES_X, cuda_TILES_Y);
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);  // 32 x 32
    //int warps_per_grid = cuda_input_image.width / TILE_SIZE;

    //print_device_arch();

    tile_sum_CUDA <<<blocksPerGrid, threadsPerBlock >>>(d_input_image_data, d_mosaic_sum, cuda_TILES_X, cuda_TILES_Y, cuda_input_image_width, cuda_input_image_height, cuda_input_image_channels);


    // Optionally during development call the skip function with the correct inputs to skip this stage
    // skip_tile_sum(input_image, mosaic_sum);

#ifdef VALIDATION
    // TODO: Uncomment and call the validation function with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    // validate_tile_sum(&input_image, mosaic_sum);
#endif
}
void cuda_stage2(unsigned char* output_global_average) {
    // Optionally during development call the skip function with the correct inputs to skip this stage
    // skip_compact_mosaic(TILES_X, TILES_Y, mosaic_sum, compact_mosaic, global_pixel_average);

#ifdef VALIDATION
    // TODO: Uncomment and call the validation functions with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    // validate_compact_mosaic(TILES_X, TILES_Y, mosaic_sum, mosaic_value, output_global_average);
#endif    
}
void cuda_stage3() {
    // Optionally during development call the skip function with the correct inputs to skip this stage
    // skip_broadcast(input_image, compact_mosaic, output_image);

#ifdef VALIDATION
    // TODO: Uncomment and call the validation function with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    // validate_broadcast(&input_image, mosaic_value, &output_image);
#endif    
}
void cuda_end(Image *output_image) {
    // This function matches the provided cuda_begin(), you may change it if desired

    // Store return value
    output_image->width = cuda_input_image.width;
    output_image->height = cuda_input_image.height;
    output_image->channels = cuda_input_image.channels;
    CUDA_CALL(hipMemcpy(output_image->data, d_output_image_data, output_image->width * output_image->height * output_image->channels * sizeof(unsigned char), hipMemcpyDeviceToHost));
    // Release allocations
    free(cuda_input_image.data);
    CUDA_CALL(hipFree(d_mosaic_value));
    CUDA_CALL(hipFree(d_mosaic_sum));
    CUDA_CALL(hipFree(d_input_image_data));
    CUDA_CALL(hipFree(d_output_image_data));
}
